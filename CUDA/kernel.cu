#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#define THREAD 16

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}


__global__ void mandelKernel(float lowerX, float lowerY, int* Pd, float stepX, float stepY, int width, int height) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    // calculate thisX, thisY
    int idx, idy;
    idx = blockIdx.x * blockDim.x + threadIdx.x; //thread block
    idy = blockIdx.y * blockDim.y + threadIdx.y; //thread block
    
    if(idx >= resX || idy >= resY)return;
    float x = lowerX + idx * stepX;
    float y = lowerY + idy * stepY;

    int index = idy * resX + idx;
    Pd[index] = mandel(x, y);
   
}

// Host front-end function that allocates the memory and launches the GPU kernel

//x1, y1, x0, y0, output, width, height, maxIterations

void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int width, int height)
{
    float stepX = (upperX - lowerX) / width;
    float stepY = (upperY - lowerY) / height;

    int blocksNumX = (int) ceil(width / (float) THREAD);
    int blocksNumY = (int) ceil(height / (float) THREAD);

    int *Pd;
    int size = width * height * sizeof(int); // image size

    hipMalloc(&Pd, size); 
    dim3 threadPerBlock(THREAD, THREAD);
    dim3 numBlocks(blocksNumX, blocksNumY);
    mandelKernel <<< numBlocks, threadPerBlock >>> (lowerX, lowerY, Pd, stepX, stepY, width, height);
    
    hipMemcpy(img, Pd, size, hipMemcpyDeviceToHost);
    
    hipFree(Pd);
}
