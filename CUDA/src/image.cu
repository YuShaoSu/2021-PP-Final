#include "hip/hip_runtime.h"
#include "image.hpp"

void Image::image_load() 
{
	img = imread(input_file);
	if(img.empty())
	{
		cout << "file open error " << endl;
	}
    height = img.rows;
	width = img.cols;
	cout << "width: " << width << ", height: "<< height << endl;
	
	pixels = new Pixel[width * height];

	// Read pixels
	for(int i = 0; i < height; i++){
		for(int j = 0; j < width; j++){
			Vec3b &color = img.at<Vec3b>(i,j);//BGR
			pixels[i*width+j].R = color[2];
			pixels[i*width+j].G = color[1];
			pixels[i*width+j].B = color[0];
		}	
	}
}


void Image::check()
{
	for(int i = 0; i < height; i++)
		for(int j = 0; j < width; j++)
			cout << "r: "<< pixels[i * width + j].R << " g: "<< pixels[i * width + j].G << " b: " << pixels[i * width + j].B;
}



__global__ void cudaGaussianFliter(Pixel* input_cuda, Pixel* output_cuda, double* matrix, const int radius, const int width, const int height) 
{
    
    int col = blockIdx.x * blockDim.x + threadIdx.x; //thread block
    int row = blockIdx.y * blockDim.y + threadIdx.y; //thread block

	if(col< width && row < height)
	{	
		Pixel res;
		res.R = res.G = res.B = 0;
		double fil;
		if (row >= radius && col >= radius && col < width - radius && row < height - radius)
		{

			for(int i = -radius; i <= radius; i++) 
				for(int j = -radius; j <= radius; j++) 
				{
					fil = matrix[(i + radius) * radius + j + radius];
					res.R += fil * input_cuda[(row + i) * width + col + j].R;
					res.G += fil * input_cuda[(row + i) * width + col + j].G;
					res.B += fil * input_cuda[(row + i) * width + col + j].B;
					
				}
		}

		output_cuda[row * width + col] = res;
	}
}


void Image::GaussianFliter(const GaussianKernel& kernel)
{

    Pixel *input_cuda, *output_cuda;
	Pixel *output = new Pixel[width * height];
	double * matrix;

    int size = width * height * sizeof(Pixel); // image size byte
	int kernel_size = kernel.dim * kernel.dim * sizeof(double);

    hipMalloc(&input_cuda, size); 
	hipMalloc(&output_cuda, size); 
	hipMalloc(&matrix, kernel_size);

	hipMemcpy(input_cuda, pixels, size, hipMemcpyHostToDevice);
	hipMemcpy(output_cuda, output, size, hipMemcpyHostToDevice);
	hipMemcpy(matrix, kernel.matrix, kernel_size, hipMemcpyHostToDevice);
	
	int block = (int) sqrt(THREAD);
    int blocksNumX = (width + block - 1) /  block;
    int blocksNumY = (height + block - 1) / block;

    dim3 threadPerBlock(block, block);
    dim3 numBlocks(blocksNumX, blocksNumY);
    cudaGaussianFliter <<< numBlocks, threadPerBlock >>> (input_cuda, output_cuda, matrix, kernel.radius, width, height);

    hipMemcpy(output, output_cuda, size, hipMemcpyDeviceToHost);
	
	memcpy(pixels, output, size);

	delete [] output;
    hipFree(input_cuda);
	hipFree(output_cuda);
	hipFree(matrix);
}



void Image::image_write()
{

	for(int i = 0; i < height; i++){
		for(int j = 0; j < width; j++){
			Vec3b &color = img.at<Vec3b>(i,j);//BGR
			color[0] = pixels[i*width+j].B;
			color[1] = pixels[i*width+j].G;
			color[2] = pixels[i*width+j].R;
		}	
	}
    
	imwrite(output_file, img);

}
	



